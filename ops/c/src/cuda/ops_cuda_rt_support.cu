#include "hip/hip_runtime.h"
/*
* Open source copyright declaration based on BSD open source template:
* http://www.opensource.org/licenses/bsd-license.php
*
* This file is part of the OPS distribution.
*
* Copyright (c) 2013, Mike Giles and others. Please see the AUTHORS file in
* the main source directory for a full list of copyright holders.
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
* * Redistributions of source code must retain the above copyright
* notice, this list of conditions and the following disclaimer.
* * Redistributions in binary form must reproduce the above copyright
* notice, this list of conditions and the following disclaimer in the
* documentation and/or other materials provided with the distribution.
* * The name of Mike Giles may not be used to endorse or promote products
* derived from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY Mike Giles ''AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
* WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL Mike Giles BE LIABLE FOR ANY
* DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
* (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
* LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
* ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

/** @brief ops cuda specific runtime support functions
  * @author Gihan Mudalige
  * @details Implements cuda backend runtime support functions
  */

//
// header files
//

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>

#include <ops_cuda_rt_support.h>
#include <ops_lib_core.h>

#include <vector>
#include <limits.h>

int ops_cyclic = 0;
/*__global__ void copy_kernel(char *dest, char *src, int size ) {
  int tid = blockIdx.x;
  memcpy(&dest[tid],&src[tid],size);
}*/

__global__ void copy_kernel_tobuf(char *dest, char *src, int rx_s, int rx_e,
                                  int ry_s, int ry_e, int rz_s, int rz_e,
                                  int x_step, int y_step, int z_step,
                                  int size_x, int size_y, int size_z,
                                  int buf_strides_x, int buf_strides_y,
                                  int buf_strides_z, int elem_size) {

  int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
  int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
  int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

  if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
      (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
      (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

    src += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * elem_size;
    dest += ((idx_z - rz_s) * z_step * buf_strides_z +
             (idx_y - ry_s) * y_step * buf_strides_y +
             (idx_x - rx_s) * x_step * buf_strides_x) *
            elem_size;
    memcpy(dest, src, elem_size);
  }
}

__global__ void copy_kernel_frombuf(char *dest, char *src, int rx_s, int rx_e,
                                    int ry_s, int ry_e, int rz_s, int rz_e,
                                    int x_step, int y_step, int z_step,
                                    int size_x, int size_y, int size_z,
                                    int buf_strides_x, int buf_strides_y,
                                    int buf_strides_z, int elem_size) {

  int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
  int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
  int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

  if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
      (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
      (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

    dest += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * elem_size;
    src += ((idx_z - rz_s) * z_step * buf_strides_z +
            (idx_y - ry_s) * y_step * buf_strides_y +
            (idx_x - rx_s) * x_step * buf_strides_x) *
           elem_size;
    memcpy(dest, src, elem_size);
  }
}

void ops_halo_copy_tobuf(char *dest, int dest_offset, ops_dat src, int rx_s,
                         int rx_e, int ry_s, int ry_e, int rz_s, int rz_e,
                         int x_step, int y_step, int z_step, int buf_strides_x,
                         int buf_strides_y, int buf_strides_z) {

  dest += dest_offset;
  int thr_x = abs(rx_s - rx_e);
  int blk_x = 1;
  if (abs(rx_s - rx_e) > 8) {
    blk_x = (thr_x - 1) / 8 + 1;
    thr_x = 8;
  }
  int thr_y = abs(ry_s - ry_e);
  int blk_y = 1;
  if (abs(ry_s - ry_e) > 8) {
    blk_y = (thr_y - 1) / 8 + 1;
    thr_y = 8;
  }
  int thr_z = abs(rz_s - rz_e);
  int blk_z = 1;
  if (abs(rz_s - rz_e) > 8) {
    blk_z = (thr_z - 1) / 8 + 1;
    thr_z = 8;
  }

  dim3 grid(blk_x, blk_y, blk_z);
  dim3 tblock(thr_x, thr_y, thr_z);
  copy_kernel_tobuf<<<grid, tblock>>>(
      dest, src->data_d, rx_s, rx_e, ry_s, ry_e, rz_s, rz_e, x_step, y_step,
      z_step, src->size[0], src->size[1], src->size[2], buf_strides_x,
      buf_strides_y, buf_strides_z, src->elem_size);

  // TODO: MPI buffers and GPUDirect
}

void ops_halo_copy_frombuf(ops_dat dest, char *src, int src_offset, int rx_s,
                           int rx_e, int ry_s, int ry_e, int rz_s, int rz_e,
                           int x_step, int y_step, int z_step,
                           int buf_strides_x, int buf_strides_y,
                           int buf_strides_z) {

  src += src_offset;
  int thr_x = abs(rx_s - rx_e);
  int blk_x = 1;
  if (abs(rx_s - rx_e) > 8) {
    blk_x = (thr_x - 1) / 8 + 1;
    thr_x = 8;
  }
  int thr_y = abs(ry_s - ry_e);
  int blk_y = 1;
  if (abs(ry_s - ry_e) > 8) {
    blk_y = (thr_y - 1) / 8 + 1;
    thr_y = 8;
  }
  int thr_z = abs(rz_s - rz_e);
  int blk_z = 1;
  if (abs(rz_s - rz_e) > 8) {
    blk_z = (thr_z - 1) / 8 + 1;
    thr_z = 8;
  }

  dim3 grid(blk_x, blk_y, blk_z);
  dim3 tblock(thr_x, thr_y, thr_z);
  copy_kernel_frombuf<<<grid, tblock>>>(
      dest->data_d, src, rx_s, rx_e, ry_s, ry_e, rz_s, rz_e, x_step, y_step,
      z_step, dest->size[0], dest->size[1], dest->size[2], buf_strides_x,
      buf_strides_y, buf_strides_z, dest->elem_size);
  dest->dirty_hd = 2;
}


__global__ void toucher(char *dat, int size, double fac) {
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  if (id < size) {
    char val = dat[id];
    if (fac == 0) val = val + 1;
    if (fac > 1) val = val * fac;
    else val = val * (char)fac;
    dat[id] = val;
  }
}
extern "C" {
void ops_touch(char *dat, int size, double fac) {
  int nthreads = 1024;
  int nblocks = (size-1)/nthreads+1;
  toucher<<<nblocks,nthreads>>>(dat,size,fac);
}
}

int mod(int a, int b)
{
    int r = a % b;
    return r < 0 ? r + b : r;
}

inline int intersection(int range1_beg, int range1_end, int range2_beg,
                 int range2_end, int *intersect_begin) {
  if (range1_beg >= range1_end || range2_beg >= range2_end) return 0;
  int i_min = MAX(range1_beg, range2_beg);
  int i_max = MIN(range1_end, range2_end);
  *intersect_begin = i_min;
  return i_max > i_min ? i_max - i_min : 0;
}


struct trans {
  int ID;
  ops_dat dat;
  int tile;
  int slot;
  int range_beg;
  int range_end;
  long hptr_begin;
  long hptr_end;
  long dptr_begin;
  long dptr_end;
};
#define E_UP 0
#define E_DOWN 1
#define E_COMP 2
#include <vector>
std::vector<trans> uploads(0);
std::vector<trans> downloads(0);
std::vector<trans> compute(0);
int upctr = 0;
int downctr = 10000;
int compctr = 20000;

void add_trans_entry(int type, int ID, ops_dat dat, int tile, int slot, int range_beg, int range_end, long hptr_begin, long hptr_end, long dptr_begin, long dptr_end) {
  trans t;
  t.ID = ID;
  if (type == E_UP && ID>=10000) printf("Error transaction log overflow\n");
  if (type == E_DOWN && ID>=20000) printf("Error transaction log overflow\n");
  t.dat = dat;
  t.tile = tile;
  t.slot = slot;
  t.range_beg = range_beg;
  t.range_end = range_end;
  t.hptr_begin = hptr_begin;
  t.hptr_end = hptr_end;
  t.dptr_begin = dptr_begin;
  t.dptr_end = dptr_end;
  if (type == E_UP)
    uploads.push_back(t);
  else if (type == E_DOWN)
    downloads.push_back(t);
  else
    compute.push_back(t);
}

void remove_trans(int ID, int type) {
  std::vector<trans>::iterator i;
  if (type == E_UP) {
    for (i = uploads.begin(); i != uploads.end();) {
      if (i->ID == ID) i = uploads.erase(i);
      else ++i;
    }
  } else if (type == E_DOWN) {
    for (i = downloads.begin(); i != downloads.end();) {
      if (i->ID == ID) i = downloads.erase(i);
      else ++i;
    }
  } else {
    for (i = compute.begin(); i != compute.end();) {
      if (i->ID == ID) i = compute.erase(i);
      else ++i;
    }
  }
}

void check_trans(int ID, int type) {
  std::vector<trans>& from = (type == E_UP ? uploads : (type==E_DOWN ? downloads : compute));

  for (int i = 0; i < from.size(); i++) {
    for (int arr = 0; arr < 3; arr++) {
      std::vector<trans>& to = (arr == E_UP ? uploads : (arr==E_DOWN ? downloads : compute));
      for (int j = 0; j < to.size(); j++) {
        int intersect_begin;
        int intersect_len = intersection(from[i].dptr_begin,from[i].dptr_end,
            to[j].dptr_begin,to[j].dptr_end,&intersect_begin);
        int intersect_len2 = intersection(from[i].hptr_begin,from[i].hptr_end,
            to[j].hptr_begin,to[j].hptr_end,&intersect_begin);
        if (from[i].ID == ID && from[i].ID != to[j].ID && from[i].dat->index == to[j].dat->index && (intersect_len > 0 || intersect_len2 > 0)) {
          printf("Error: %s new %d (%d) slot %d intersecting with old %d (%d) slot %d: device %ld-%ld vs %ld-%ld host %ld-%ld vs %ld-%ld\n",
              from[i].dat->name, type, ID, from[i].slot,
              arr, to[j].ID, to[j].slot, 
              from[i].dptr_begin,from[i].dptr_end,
                to[j].dptr_begin,to[j].dptr_end,
              from[i].hptr_begin,from[i].hptr_end,
                to[j].hptr_begin,to[j].hptr_end);
        }
      }
    }
  }
}

struct datasets {
  int size[OPS_MAX_DIM];
  int base_offset;
  int max_width;
  int curr_slot;
  int curr_chunk[2];
  int actually_uploaded;
  ops_dat dat;
  long bytes;
  long curr_offset;
  long last_offset;
  long curr_size;
  long last_size;
  long copy_from;
  long copy_amount;
};

std::vector<datasets> dats(0);

hipStream_t stream_copy_up = 0;
hipStream_t stream_copy_down = 0;
hipStream_t stream_compute = 0;
hipStream_t stream = 0;

int upload_me(int idx) {
  //return !(idx>=5 && idx <= 31);
  //return !((idx>=5 && idx <= 31) || idx <=-1 || idx == 1 || idx == 3);
  return 0;
}

//TODO: v1 seems okay 
void ops_get_offsets_deprange(long &base_ptr, long &end_ptr, ops_dat dat, std::vector<std::vector<int> > &dependency_ranges, int tile, int num_tiles, int lrf, long &delta) {

  //calculate base_offset without last used dimension
  long line_begin_offset = 0;
  long cumsize = 1;
  for (int i = 0; i < dat->block->dims-1; i++) {
    line_begin_offset +=
        dat->elem_size * cumsize * (-dat->base[i] - dat->d_m[i]); //TODO: different for MPI
    cumsize *= dats[dat->index].size[i];
  }
  line_begin_offset = dats[dat->index].base_offset - line_begin_offset;
  base_ptr = line_begin_offset; //go back to beginning of x line in 2D or last x-y plane in 3D
  end_ptr = line_begin_offset; //go back to beginning of x line in 2D or last x-y plane in 3D
  //For first n-1 dimensions, we copy all of them
  long prod = dat->elem_size;
  for (int d = 0; d < dat->block->dims-1; d++) {
    prod *= dats[dat->index].size[d];
  }
  //For the nth dimension we just copy up to where we need it
  int d = dat->block->dims -1;
  int prevrange = dependency_ranges[dat->index][mod(tile-1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 1] - dependency_ranges[dat->index][mod(tile-1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 0];
  int nextrange = dependency_ranges[dat->index][mod(tile+1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 1] - dependency_ranges[dat->index][mod(tile+1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 0];
  //Left or full - start of this tile
  if (lrf == 0 || lrf == 2 || tile == 0 || dat->size[d] == 1 || prevrange == 0)
    base_ptr += dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0] * prod;
  else //right - end of previous tile
    base_ptr += MAX(dependency_ranges[dat->index][mod(tile-1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 1],
                    dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0]) * prod;
  //Right or full - end of this tile
  if (lrf == 1 || lrf ==2 || tile == num_tiles-1 || dat->size[d] == 1 || nextrange == 0) 
    end_ptr  += dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1] * prod;
  else //left - start of next tile
    end_ptr  += MIN(dependency_ranges[dat->index][mod(tile+1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 0],
                    dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1]) * prod;

  if (dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0] 
      == dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1]) end_ptr = base_ptr; //zero dependency

  //I need to offset the beginning fo tile 0, so tile 3 (laoding to slot 0) won't bite tile 1's tail
  //extra space needed is the difference between largest tile and left range of tile 0
  if (num_tiles > 1 && tile == 0 && nextrange > 1 && dependency_ranges[dat->index][0 * 2 * OPS_MAX_DIM + 2 * d + 1] - dependency_ranges[dat->index][0 * 2 * OPS_MAX_DIM + 2 * d + 0] > 1)
    delta = MAX(2,(dats[dat->index].max_width - dependency_ranges[dat->index][1 * 2 * OPS_MAX_DIM + 2 * d + 0] + dependency_ranges[dat->index][0 * 2 * OPS_MAX_DIM + 2 * d + 0]))*prod;
  else delta = 0;

  if (end_ptr < base_ptr) {printf("WARNING: overreaching depranges! Please check, shouldn't happen\n%s %ld-%ld, dep range: %d-%d prev %d - %d next start %d\n",dat->name, base_ptr, end_ptr, dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0], dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1],dependency_ranges[dat->index][mod(tile-1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 0],dependency_ranges[dat->index][mod(tile-1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 1], dependency_ranges[dat->index][mod(tile+1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 0]); end_ptr = base_ptr;} //zero ranges
}

void ops_tiling_init_streams() {
  cutilSafeCall(hipStreamCreateWithFlags(&stream_copy_down,hipStreamNonBlocking));
  cutilSafeCall(hipStreamCreateWithFlags(&stream_copy_up,hipStreamNonBlocking));
  int leastPriority, greatestPriority;
  hipDeviceGetStreamPriorityRange ( &leastPriority, &greatestPriority );
  cutilSafeCall(hipStreamCreateWithPriority(&stream_compute,hipStreamNonBlocking,greatestPriority));
  stream = stream_compute;
}

void ops_tiling_datastructures_init(int size) {
  dats.resize(size);
  for (int i = 0; i < dats.size(); i++) {
    dats[i].bytes = 0;
    dats[i].dat = NULL;
  }
  ops_dat_entry *item, *tmp_item;
  for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
    tmp_item = TAILQ_NEXT(item, entries);
    dats[item->dat->index].dat = item->dat;
    dats[item->dat->index].base_offset = item->dat->base_offset;
    memcpy(dats[item->dat->index].size, item->dat->size, sizeof(int)*OPS_MAX_DIM);
  }
}

int deviceIdUM = -1;
hipEvent_t e1, e2;

void ops_prepare_tile_managed(int tile, int total_tiles, std::vector<std::vector<int> > &tiled_ranges, std::vector<std::vector<int> > &dependency_ranges, std::vector<int> &datasets_access_type) {
  int first = 0;
  //First time
  if (deviceIdUM == -1) {
    first = 1;
    cutilSafeCall(hipGetDevice(&deviceIdUM));
    ops_tiling_init_streams();
    ops_tiling_datastructures_init(dependency_ranges.size());
    hipEventCreate(&e1);
    hipEventCreate(&e2);
  }
  hipEventSynchronize(e1);
  hipEventSynchronize(e2);
  if (tile == 0 && first) {
    ops_dat_entry *item, *tmp_item;
    for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
      tmp_item = TAILQ_NEXT(item, entries);
      ops_dat dat = item->dat;
      long base_ptr, end_ptr, delta;
      ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, tile, total_tiles, 2, delta); //Full
      if (end_ptr > base_ptr)
        cutilSafeCall(hipMemPrefetchAsync(dat->data+base_ptr,end_ptr-base_ptr,deviceIdUM,stream));
    }
    cutilSafeCall(hipStreamSynchronize(stream));
  }
}

void ops_finish_tile_managed(int tile, int total_tiles, std::vector<std::vector<int> > &tiled_ranges, std::vector<std::vector<int> > &dependency_ranges, std::vector<int> &datasets_access_type) {
  hipEventRecord(e1, stream);
  int next_tile = (tile+1)%total_tiles;
  int prev_tile = mod(tile-1,total_tiles);
  ops_dat_entry *item, *tmp_item;
  for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
    tmp_item = TAILQ_NEXT(item, entries);
    ops_dat dat = item->dat;
    hipStreamSynchronize(stream_copy_up);
    {
      long base_ptr, end_ptr, delta;
      ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, next_tile, total_tiles, next_tile == 0 ? 2 : 1, delta); //Right
      if (end_ptr > base_ptr)
        cutilSafeCall(hipMemPrefetchAsync(dat->data+base_ptr,end_ptr-base_ptr,deviceIdUM,stream_copy_up));
      else {
        long base_ptr, end_ptr, delta;
        ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, (next_tile+1)%total_tiles, total_tiles, (next_tile+1)%total_tiles == 0 ? 2 : 1, delta); //Right
        if (end_ptr > base_ptr)
          cutilSafeCall(hipMemPrefetchAsync(dat->data+base_ptr,end_ptr-base_ptr,deviceIdUM,stream_copy_up));
      }
    }
      long base_ptr, end_ptr, delta;
      //ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, tile, total_tiles, tile == total_tiles -1 ? 2 : 0, delta); //Left
      ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, prev_tile, total_tiles, prev_tile == total_tiles -1 ? 2 : 0, delta); //Left
      if (end_ptr > base_ptr)
        cutilSafeCall(hipMemPrefetchAsync(dat->data+base_ptr,end_ptr-base_ptr,hipCpuDeviceId,stream));
  }
  hipEventRecord(e2, stream_copy_up);
/*  if (tile == total_tiles-1) {
    for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
      tmp_item = TAILQ_NEXT(item, entries);
      ops_dat dat = item->dat;
      int idx = dat->index;
      hipStreamSynchronize(stream_copy_up);
      long base_ptr, end_ptr, delta;
      ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, tile, total_tiles, 2, delta); //Left
      if (end_ptr > base_ptr)
        cutilSafeCall(hipMemPrefetchAsync(dat->data+base_ptr,end_ptr-base_ptr,hipCpuDeviceId,stream));
    }
  }*/
  // rotate streams and swap events
  hipStream_t st;
  hipEvent_t et;
  st = stream; stream = stream_copy_up; stream_copy_up = st;
  st = stream_copy_up; stream_copy_up = stream_copy_down; stream_copy_down = st;
  et = e1; e1 = e2; e2 = et;
}

void ops_prepare_tile(int tile, int total_tiles, std::vector<std::vector<int> > &tiled_ranges, std::vector<std::vector<int> > &dependency_ranges, std::vector<int> &datasets_access_type) {

  if (ops_managed) {ops_prepare_tile_managed(tile, total_tiles, tiled_ranges, dependency_ranges, datasets_access_type); return;}
  if (tile == 0) {
    cutilSafeCall(hipStreamSynchronize(stream_copy_up));
    cutilSafeCall(hipStreamSynchronize(stream_compute));
//    remove_trans(compctr,E_COMP);
//    remove_trans(upctr,E_UP);

    //First time ever - initialise
    if (dats.size()==0) {
      ops_tiling_datastructures_init(dependency_ranges.size());
      ops_tiling_init_streams();
    }

    //determine biggest dependency range for each dataset to allocate scratch memory on GPU
    ops_dat_entry *item, *tmp_item;
    for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
      tmp_item = TAILQ_NEXT(item, entries);
      int idx = item->dat->index;
      int maxsize = 0;
      for (int t = 0; t < total_tiles; t++) {
        int d = item->dat->block->dims-1; //Only for last dimension
        maxsize = MAX(maxsize,dependency_ranges[idx][t * 2 * OPS_MAX_DIM + 2 * d + 1]
                            - dependency_ranges[idx][t * 2 * OPS_MAX_DIM + 2 * d + 0]);
      }
      dats[idx].max_width = maxsize;
      //Allocate it a little larger, if not edge dat in this dim (or just unused)
      if (maxsize > 1 && dats[idx].bytes == 0) maxsize += 17; 

      // total required memory
      long cum_size = item->dat->elem_size; 
      for (int d = 0; d < item->dat->block->dims-1; d++) cum_size *= item->dat->size[d];
      cum_size *= maxsize;

      //3 slots
      cum_size *= 3;
      if (cum_size > dats[idx].bytes) {
        //printf("Reallocating memory for %s: %ld->%ld\n",item->dat->name, dats[idx].bytes,cum_size);
        cutilSafeCall(hipStreamSynchronize(stream_copy_down)); //Need to make sure all previous copies finished before dealloc
        cutilSafeCall(hipFree(item->dat->data_d));
        cutilSafeCall(hipMalloc(&item->dat->data_d, cum_size));
        dats[idx].bytes = cum_size;
        dats[idx].curr_slot = 2; //last used slot, so next one is 0
        dats[idx].curr_chunk[0] = 0;
        dats[idx].curr_chunk[1] = 0;
        dats[idx].curr_offset = 0;
        dats[idx].last_offset = 0;
        dats[idx].last_size = 0;
        dats[idx].curr_size = 0;
        dats[idx].copy_from = 0;
        dats[idx].copy_amount = 0;
        dats[idx].actually_uploaded = 0;
      }
    }
  }

  //Create event that we can sync on an the end to make sure previous copies have finished 
  hipEvent_t e_copyup;
  hipEventCreate(&e_copyup);
  if (tile != 0)
    hipEventRecord(e_copyup, stream_copy_up);

  //Copy required data to GPU
  //TODO: assert that only last dim is tiled
  //TODO: SoA: set dat->size[dat->block->dims-1]
  ops_dat_entry *item, *tmp_item;
  if (tile == 0) {
//#define NOPREFETCH
#ifdef NOPREFETCH
    upctr++; 
#endif
    for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
      tmp_item = TAILQ_NEXT(item, entries);
      ops_dat dat = item->dat;
      int idx = dat->index;

      //Determine data to required on to the GPU
      long base_ptr, end_ptr, delta;
      ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, tile, total_tiles, 2, delta); //Full
      //If no data needed
      if (end_ptr-base_ptr == 0) {
        continue;
      }
      long slice_size = (end_ptr - base_ptr) / (dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1]-
              dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0]);
      int intersect_begin;
      int intersect_len = intersection(dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0],
                                       dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1],
                                       dats[idx].curr_chunk[0],
                                       dats[idx].curr_chunk[1],&intersect_begin);
      //If already uploaded
      if (intersect_begin == dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0] && 
           intersect_len == (dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1]-
                             dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0]) &&
          //either needs upload and uploaded, or doesn't need upload
          (((datasets_access_type[idx] > 0 || (datasets_access_type[idx] == 0 && upload_me(idx))) &&  dats[idx].actually_uploaded == 1)
           ||(datasets_access_type[idx] == 0 && !upload_me(idx)) )) {

        //if the original started before this one
        if (dats[idx].curr_chunk[0] < dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0]) {
          dats[idx].curr_offset += (dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0]-dats[idx].curr_chunk[0]) * slice_size;
        }
        //if the original was longer
        dats[idx].curr_size = end_ptr-base_ptr;
        //update for this chunk
        dats[idx].curr_chunk[0] = dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0];
        dats[idx].curr_chunk[1] = dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1];
        continue;
      };

      //If no intersection, start upload into next slot
      if (intersect_len == 0
#ifdef NOPREFETCH
                    || true
#endif
                    ) {
        dats[idx].curr_slot = mod(dats[idx].curr_slot+1,3);
        dats[idx].last_offset = dats[idx].curr_offset;
        dats[idx].curr_offset = (dats[idx].bytes/3) * dats[idx].curr_slot;
        if (dats[idx].curr_offset + end_ptr-base_ptr > (dats[idx].bytes/3) * (dats[idx].curr_slot+1)) printf("Error, out of bounds copy for %s in tile==0: copying tile %d to slot %d: %p+%ld size %ld, but size is %ld\n",dat->name, tile, dats[idx].curr_slot, dat->data_d, dats[idx].curr_offset, end_ptr - base_ptr, (dats[idx].bytes/3) * (dats[idx].curr_slot+1));
        if (datasets_access_type[idx] > 0 || (datasets_access_type[idx] == 0 && upload_me(idx))) { //read first 
          //printf("Tile 0 fetching to NEW slot %d Copying %s from %p+%ld to %p+%ld (%p-%p), size %ld, delta %ld\n", dats[idx].curr_slot, dat->name, dat->data, base_ptr, dat->data_d,dats[idx].curr_offset, dat->data_d, dat->data_d + dats[idx].bytes, end_ptr-base_ptr,delta);
          dats[idx].actually_uploaded = 1;
          cutilSafeCall(hipMemcpyAsync(dat->data_d+dats[idx].curr_offset, dat->data + base_ptr, end_ptr - base_ptr, hipMemcpyHostToDevice, stream_copy_up));
//          add_trans_entry(E_UP, upctr, dat, tile, dats[idx].curr_slot,
//              dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0],
//              dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1],
//              base_ptr, end_ptr, dats[idx].curr_offset, dats[idx].curr_offset+end_ptr-base_ptr);
        } else dats[idx].actually_uploaded = 0;
      //if not actually uploaded 
      } else if (dats[idx].actually_uploaded == 0 && (datasets_access_type[idx] > 0 || (datasets_access_type[idx] == 0 && upload_me(idx)))) {
          dats[idx].actually_uploaded = 1;
          cutilSafeCall(hipMemcpyAsync(dat->data_d+dats[idx].curr_offset, dat->data + base_ptr, end_ptr - base_ptr, hipMemcpyHostToDevice, stream_copy_up));
//          add_trans_entry(E_UP, upctr, dat, tile, dats[idx].curr_slot,
//              dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0],
//              dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1],
//              base_ptr, end_ptr, dats[idx].curr_offset, dats[idx].curr_offset+end_ptr-base_ptr);
      } //if partly uploaded
      else {
        //Missing some in the beginning
        if (dats[idx].curr_chunk[0] > dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0]) {
          long extra_data = slice_size * (dats[idx].curr_chunk[0] - dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0]);
          dats[idx].curr_offset -= extra_data;
          dats[idx].curr_size += extra_data;
          if (dats[idx].curr_offset < 0 || (dats[idx].curr_slot>0 && dats[idx].curr_offset < (dats[idx].last_offset + dats[idx].last_size)))
            printf("Error: missing left side of tile 0 overwriting previous tile or offset < 0\n");
          if (datasets_access_type[idx] > 0 || (datasets_access_type[idx] == 0 && upload_me(idx))) //read first
            cutilSafeCall(hipMemcpyAsync(dat->data_d+dats[idx].curr_offset, dat->data + base_ptr, extra_data, hipMemcpyHostToDevice, stream_copy_up));
        }
        //Missing some in the end
        if (dats[idx].curr_chunk[1] < dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1]) {
          //if the speculatively prefetched started before this one
          if (dats[idx].curr_chunk[0] < dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0]) {
            dats[idx].curr_offset += (dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0]-dats[idx].curr_chunk[0]) * slice_size;
            dats[idx].curr_size -= (dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0]-dats[idx].curr_chunk[0]) * slice_size;
          }
          long extra_data = slice_size * (dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1] - dats[idx].curr_chunk[1]);          
          if (dats[idx].curr_offset + (end_ptr - base_ptr) > (dats[idx].bytes/3) * (dats[idx].curr_slot+1)) 
            printf("Error: missing right side too large\n");
          if (datasets_access_type[idx] > 0 || (datasets_access_type[idx] == 0 && upload_me(idx))) //read first
            cutilSafeCall(hipMemcpyAsync(dat->data_d+dats[idx].curr_offset+dats[idx].curr_size, dat->data + end_ptr - extra_data, extra_data, hipMemcpyHostToDevice, stream_copy_up));
        }
      }
      dats[idx].last_size = dats[idx].curr_size;
      dats[idx].curr_size = end_ptr-base_ptr;
      dats[idx].curr_chunk[0] = dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0];
      dats[idx].curr_chunk[1] = dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1];
      dats[idx].copy_from = 0;
      dats[idx].copy_amount = 0;
    }
    hipEventRecord(e_copyup, stream_copy_up);
//    check_trans(upctr,E_UP);

  }
#ifdef NOPREFETCH
  if (tile != total_tiles-1) {
  if (tile == 0) {cutilSafeCall(hipStreamSynchronize(stream_copy_down)); /*remove_trans(downctr,E_DOWN);*/}
#endif
  upctr++;
  //upload next tile - leave a few rows spare if next tile's dependency range is bigger
  for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
    tmp_item = TAILQ_NEXT(item, entries);
    ops_dat dat = item->dat;
    int idx = dat->index;

    //For edge thats I do not need to upload again
    //TODO: v1
    if (dat->size[dat->block->dims-1] == 1) continue;

    //Upload next tile
    int next_tile = mod(tile+1,total_tiles);
    long base_ptr, end_ptr, delta;
    ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, next_tile, total_tiles, next_tile == 0 ? 2 : 1, delta); //Right
    if (end_ptr-base_ptr == 0) {
      continue;
    }
    //
    dats[idx].curr_slot = mod(dats[idx].curr_slot+1,3);
    //Compute Full range, then right begin - full begin is the extra offset
    long base_ptr2, end_ptr2;
    ops_get_offsets_deprange(base_ptr2, end_ptr2, dat, dependency_ranges, next_tile, total_tiles, 2, delta); //Full
    long extra_offset = base_ptr - base_ptr2;
    dats[idx].last_offset = dats[idx].curr_offset;
    if (next_tile == 0) {
      long slice_size = (end_ptr - base_ptr) / (dependency_ranges[idx][next_tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1]-
          dependency_ranges[idx][next_tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0]);
      dats[idx].curr_offset = (dats[idx].bytes/3) * dats[idx].curr_slot + 2*slice_size;
    } else
      dats[idx].curr_offset = (dats[idx].bytes/3) * dats[idx].curr_slot;
    dats[idx].last_size = dats[idx].curr_size;
    dats[idx].curr_size = end_ptr - base_ptr2; //Full size
    if (dats[idx].curr_offset + end_ptr-base_ptr2 > (dats[idx].bytes/3)*(dats[idx].curr_slot+1)) printf("Error, out of bounds copy for %s: copying tile %d to slot %d: %p+%ld size %ld, but size is %ld\n",dat->name, next_tile, dats[idx].curr_slot, dat->data_d, dats[idx].curr_offset, end_ptr - base_ptr2, (dats[idx].bytes/3)*(dats[idx].curr_slot+1));
    if (datasets_access_type[idx] > 0 || (datasets_access_type[idx] == 0 && upload_me(idx))) { //read first 
      //printf("Prefetching tile %d to slot %d Copying %s from %p+%ld to %p+%ld (%p-%p), size %ld, delta %ld\n", next_tile, dats[idx].curr_slot, dat->name, dat->data, base_ptr, dat->data_d, dats[idx].curr_offset + extra_offset, dat->data_d, dat->data_d + dats[idx].bytes, end_ptr-base_ptr, 0);
      dats[idx].actually_uploaded = 1;
      cutilSafeCall(hipMemcpyAsync(dat->data_d + dats[idx].curr_offset + extra_offset,
            dat->data + base_ptr, end_ptr - base_ptr, hipMemcpyHostToDevice, stream_copy_up));
//      add_trans_entry(E_UP, upctr, dat, next_tile, 0,
//          dependency_ranges[idx][next_tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0],
//          dependency_ranges[idx][next_tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1],
//          base_ptr, end_ptr, dats[idx].curr_offset + extra_offset, dats[idx].curr_offset+end_ptr-base_ptr);
    } else dats[idx].actually_uploaded = 0;
    if (next_tile == 0) { //Speculative prefetch
      dats[idx].copy_from = 0;
      dats[idx].copy_amount = 0;
    } else {
      dats[idx].copy_from = dats[idx].last_offset + dats[idx].last_size - extra_offset;
      dats[idx].copy_amount = extra_offset;
    }
    dats[idx].curr_chunk[0] = dependency_ranges[dat->index][next_tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0];
    dats[idx].curr_chunk[1] = dependency_ranges[dat->index][next_tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1];
  }
//  check_trans(upctr,E_UP); 
#ifdef NOPREFETCH
    }
#endif

  compctr++;
  //alter base_offset so that it is offset by the dependency range for the current tile
  for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
    tmp_item = TAILQ_NEXT(item, entries);
    ops_dat dat = item->dat;
      long base_ptr, end_ptr, delta;
      ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, tile, total_tiles, 2, delta); //Full
      dat->base_offset = dats[dat->index].base_offset - base_ptr +
#ifdef NOPREFETCH
           (tile == total_tiles-1 ? dats[dat->index].curr_offset : dats[dat->index].last_offset);
#else
           dats[dat->index].last_offset;
#endif
/*      long base_ptr, end_ptr, delta;
      ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, tile, total_tiles, 2, delta); //Full
      int curr_slot = 
#ifdef NOPREFETCH
        (tile == total_tiles-1) ? dats[dat->index].curr_slot : mod(dats[dat->index].curr_slot-1,3);
#else
        mod(dats[idx].curr_slot-1,3);
#endif
    add_trans_entry(E_COMP, compctr, dat, tile, curr_slot,
        dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0],
        dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1],
        base_ptr, end_ptr, dat->base_offset+base_ptr, dat->base_offset+end_ptr-base_ptr);*/
  }

  //Before actually starting the computations, make sure previous copies up finished
  hipEventSynchronize(e_copyup);
  hipEventDestroy(e_copyup);
//  int prev_up_idx = 
//#ifdef NOPREFETCH
//    (tile == total_tiles-1) ? upctr : (upctr - 1);
//#else
//    upctr - 1;
//#endif
//  remove_trans(prev_up_idx, E_UP);
//  check_trans(compctr,E_COMP); 
}
void ops_finish_tile(int tile, int total_tiles, std::vector<std::vector<int> > &tiled_ranges, std::vector<std::vector<int> > &dependency_ranges, std::vector<int> &datasets_access_type) {
  if (ops_managed) {ops_finish_tile_managed(tile, total_tiles, tiled_ranges, dependency_ranges, datasets_access_type); return;}
  hipEvent_t e_copydown;
  hipEventCreate(&e_copydown);
  hipEventRecord(e_copydown, stream_copy_down);

  //Wait for compute to finish before downloading results
  cutilSafeCall(hipStreamSynchronize(stream_compute));
//  remove_trans(compctr,E_COMP);
  
  downctr++;
  //TODO: do not copy back read-only data
  ops_dat_entry *item, *tmp_item;
  for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
    tmp_item = TAILQ_NEXT(item, entries);
    ops_dat dat = item->dat;
    int idx = dat->index;
    //Restore properties
    if (tile == total_tiles-1)
      dat->base_offset = dats[dat->index].base_offset;

    //Skip edge dats, those are managed by the first tile
    if (tile > 0 && dat->size[dat->block->dims-1] == 1) continue;

    //Copy over the right edge of this tile in the previous slot, to the left of the next slot
    if (dats[idx].copy_amount > 0) {
      if (dats[idx].copy_from < 0 || dats[idx].copy_amount < 0 || dats[idx].copy_from + dats[idx].copy_amount > dats[idx].bytes) printf("Error: right edge to start overreach %s: from %ld to %ld, size %ld\n",dats[idx].dat->name,dats[idx].copy_from,dats[idx].copy_from+dats[idx].copy_amount,dats[idx].bytes);
      long toptr = dats[idx].curr_offset; 
      //printf("Copying %s end->start from %ld size %ld to %ld\n",dat->name, dats[idx].copy_from, dats[idx].copy_amount, toptr);
      cutilSafeCall(hipMemcpyAsync(dat->data_d + toptr, dat->data_d+dats[idx].copy_from, dats[idx].copy_amount, hipMemcpyDeviceToDevice, stream_compute));
      dats[idx].copy_from = 0;
      dats[idx].copy_amount = 0;
    }
    //Determine data to be copied off to the CPU
    long base_ptr, end_ptr, delta;
    ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, tile, total_tiles, (tile == total_tiles-1 ? 2 : 0),delta); //Left or full if last tile
    //Where the data being processed starts
    long base_ptr_gpu = 
#ifdef NOPREFETCH
      tile == total_tiles-1 ? dats[idx].curr_offset : dats[idx].last_offset;
#else
      dats[idx].last_offset;
#endif
    if (datasets_access_type[idx] == 0 && !upload_me(idx) && ops_cyclic) continue;
    if (datasets_access_type[idx] != 1  && end_ptr-base_ptr>0) { //not read only
      //printf("Tile %d copying back %s to %p+%ld from %p+%ld, size %ld\n", tile, dat->name, dat->data, base_ptr, dat->data_d, base_ptr_gpu,end_ptr-base_ptr);
      cutilSafeCall(hipMemcpyAsync(dat->data + base_ptr, dat->data_d+base_ptr_gpu, end_ptr - base_ptr, hipMemcpyDeviceToHost, stream_copy_down));
//      int curr_slot = 
//#ifdef NOPREFETCH
//        (tile == total_tiles-1) ? dats[idx].curr_slot : mod(dats[idx].curr_slot-1,3);
//#else
//        mod(dats[idx].curr_slot-1,3);
//#endif
//      add_trans_entry(E_DOWN, downctr, dat, tile, curr_slot,
//          dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0],
//          dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1],
//          base_ptr, end_ptr, base_ptr_gpu, base_ptr_gpu+end_ptr-base_ptr);
    }
  }
//  check_trans(downctr,E_DOWN); 
  
  //Wait for previous round of copies to finish
  hipEventSynchronize(e_copydown);
  hipEventDestroy(e_copydown);
//  remove_trans(downctr-1,E_DOWN);
}

