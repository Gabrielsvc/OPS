#include "hip/hip_runtime.h"
/*
* Open source copyright declaration based on BSD open source template:
* http://www.opensource.org/licenses/bsd-license.php
*
* This file is part of the OPS distribution.
*
* Copyright (c) 2013, Mike Giles and others. Please see the AUTHORS file in
* the main source directory for a full list of copyright holders.
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
* * Redistributions of source code must retain the above copyright
* notice, this list of conditions and the following disclaimer.
* * Redistributions in binary form must reproduce the above copyright
* notice, this list of conditions and the following disclaimer in the
* documentation and/or other materials provided with the distribution.
* * The name of Mike Giles may not be used to endorse or promote products
* derived from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY Mike Giles ''AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
* WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL Mike Giles BE LIABLE FOR ANY
* DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
* (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
* LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
* ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

/** @brief ops cuda specific runtime support functions
  * @author Gihan Mudalige
  * @details Implements cuda backend runtime support functions
  */

//
// header files
//

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>

#include <ops_cuda_rt_support.h>
#include <ops_lib_core.h>

#include <vector>
#include <limits.h>

/*__global__ void copy_kernel(char *dest, char *src, int size ) {
  int tid = blockIdx.x;
  memcpy(&dest[tid],&src[tid],size);
}*/

__global__ void copy_kernel_tobuf(char *dest, char *src, int rx_s, int rx_e,
                                  int ry_s, int ry_e, int rz_s, int rz_e,
                                  int x_step, int y_step, int z_step,
                                  int size_x, int size_y, int size_z,
                                  int buf_strides_x, int buf_strides_y,
                                  int buf_strides_z, int elem_size) {

  int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
  int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
  int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

  if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
      (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
      (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

    src += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * elem_size;
    dest += ((idx_z - rz_s) * z_step * buf_strides_z +
             (idx_y - ry_s) * y_step * buf_strides_y +
             (idx_x - rx_s) * x_step * buf_strides_x) *
            elem_size;
    memcpy(dest, src, elem_size);
  }
}

__global__ void copy_kernel_frombuf(char *dest, char *src, int rx_s, int rx_e,
                                    int ry_s, int ry_e, int rz_s, int rz_e,
                                    int x_step, int y_step, int z_step,
                                    int size_x, int size_y, int size_z,
                                    int buf_strides_x, int buf_strides_y,
                                    int buf_strides_z, int elem_size) {

  int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
  int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
  int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

  if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
      (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
      (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

    dest += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * elem_size;
    src += ((idx_z - rz_s) * z_step * buf_strides_z +
            (idx_y - ry_s) * y_step * buf_strides_y +
            (idx_x - rx_s) * x_step * buf_strides_x) *
           elem_size;
    memcpy(dest, src, elem_size);
  }
}

void ops_halo_copy_tobuf(char *dest, int dest_offset, ops_dat src, int rx_s,
                         int rx_e, int ry_s, int ry_e, int rz_s, int rz_e,
                         int x_step, int y_step, int z_step, int buf_strides_x,
                         int buf_strides_y, int buf_strides_z) {

  dest += dest_offset;
  int thr_x = abs(rx_s - rx_e);
  int blk_x = 1;
  if (abs(rx_s - rx_e) > 8) {
    blk_x = (thr_x - 1) / 8 + 1;
    thr_x = 8;
  }
  int thr_y = abs(ry_s - ry_e);
  int blk_y = 1;
  if (abs(ry_s - ry_e) > 8) {
    blk_y = (thr_y - 1) / 8 + 1;
    thr_y = 8;
  }
  int thr_z = abs(rz_s - rz_e);
  int blk_z = 1;
  if (abs(rz_s - rz_e) > 8) {
    blk_z = (thr_z - 1) / 8 + 1;
    thr_z = 8;
  }

  dim3 grid(blk_x, blk_y, blk_z);
  dim3 tblock(thr_x, thr_y, thr_z);
  copy_kernel_tobuf<<<grid, tblock>>>(
      dest, src->data_d, rx_s, rx_e, ry_s, ry_e, rz_s, rz_e, x_step, y_step,
      z_step, src->size[0], src->size[1], src->size[2], buf_strides_x,
      buf_strides_y, buf_strides_z, src->elem_size);

  // TODO: MPI buffers and GPUDirect
}

void ops_halo_copy_frombuf(ops_dat dest, char *src, int src_offset, int rx_s,
                           int rx_e, int ry_s, int ry_e, int rz_s, int rz_e,
                           int x_step, int y_step, int z_step,
                           int buf_strides_x, int buf_strides_y,
                           int buf_strides_z) {

  src += src_offset;
  int thr_x = abs(rx_s - rx_e);
  int blk_x = 1;
  if (abs(rx_s - rx_e) > 8) {
    blk_x = (thr_x - 1) / 8 + 1;
    thr_x = 8;
  }
  int thr_y = abs(ry_s - ry_e);
  int blk_y = 1;
  if (abs(ry_s - ry_e) > 8) {
    blk_y = (thr_y - 1) / 8 + 1;
    thr_y = 8;
  }
  int thr_z = abs(rz_s - rz_e);
  int blk_z = 1;
  if (abs(rz_s - rz_e) > 8) {
    blk_z = (thr_z - 1) / 8 + 1;
    thr_z = 8;
  }

  dim3 grid(blk_x, blk_y, blk_z);
  dim3 tblock(thr_x, thr_y, thr_z);
  copy_kernel_frombuf<<<grid, tblock>>>(
      dest->data_d, src, rx_s, rx_e, ry_s, ry_e, rz_s, rz_e, x_step, y_step,
      z_step, dest->size[0], dest->size[1], dest->size[2], buf_strides_x,
      buf_strides_y, buf_strides_z, dest->elem_size);
  dest->dirty_hd = 2;
}


__global__ void toucher(char *dat, int size, double fac) {
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  if (id < size) {
    char val = dat[id];
    if (fac == 0) val = val + 1;
    if (fac > 1) val = val * fac;
    else val = val * (char)fac;
    dat[id] = val;
  }
}
extern "C" {
void ops_touch(char *dat, int size, double fac) {
  int nthreads = 1024;
  int nblocks = (size-1)/nthreads+1;
  toucher<<<nblocks,nthreads>>>(dat,size,fac);
}
}

struct datasets {
  long bytes;
  ops_dat dat;
  int size[OPS_MAX_DIM];
  int base_offset;
};

std::vector<datasets> dats(0);

hipStream_t stream_copy_up = 0;
hipStream_t stream_copy_down = 0;
hipStream_t stream_compute = 0;

void ops_prepare_tile(int tile, int total_tiles, std::vector<std::vector<int> > &tiled_ranges, std::vector<std::vector<int> > &dependency_ranges) {
  //Wait for previous downloads to CPU finish
  cutilSafeCall(hipStreamSynchronize(stream_copy_down));

  if (tile == 0) {
    //First time ever - initialise
    if (dats.size()==0) {
      dats.resize(dependency_ranges.size());
      for (int i = 0; i < dats.size(); i++) {
        dats[i].bytes = 0;
        dats[i].dat = NULL;
      }
      ops_dat_entry *item, *tmp_item;
      for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
        tmp_item = TAILQ_NEXT(item, entries);
        dats[item->dat->index].dat = item->dat;
        dats[item->dat->index].base_offset = item->dat->base_offset;
        memcpy(dats[item->dat->index].size, item->dat->size, sizeof(int)*OPS_MAX_DIM);
      }
    }

    //determine biggest dependency range for each dataset to allocate scratch memory on GPU
    ops_dat_entry *item, *tmp_item;
    for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
      tmp_item = TAILQ_NEXT(item, entries);
      int idx = item->dat->index;
      int maxsize[OPS_MAX_DIM]; for (int i = 0; i < OPS_MAX_DIM; i++) maxsize[i] = -INT_MAX;
      for (int t = 0; t < total_tiles; t++) {
        for (int d = 0; d < item->dat->block->dims; d++) {
          maxsize[d] = MAX(maxsize[d],dependency_ranges[idx][t * 2 * OPS_MAX_DIM + 2 * d + 1]
                                    - dependency_ranges[idx][t * 2 * OPS_MAX_DIM + 2 * d + 0]);
        }
      }

      //Allocate it a little larger
      if (maxsize[item->dat->block->dims-1] != 0 && dats[idx].bytes == 0) maxsize[item->dat->block->dims-1] += 13;

      //TODO: assure only last dim is tiled
      for (int d = 0; d < item->dat->block->dims - 1; d++) maxsize[d] = item->dat->size[d];

      // total required memory
      long cum_size = item->dat->elem_size; 
      for (int d = 0; d < item->dat->block->dims; d++) cum_size *= (maxsize[d]);
      if (cum_size > dats[idx].bytes) {
        //printf("Reallocating memory for %s: %ld->%ld\n",item->dat->name, dats[idx].bytes,cum_size);
        cutilSafeCall(hipFree(item->dat->data_d));
        cutilSafeCall(hipMalloc(&item->dat->data_d, cum_size));
        dats[idx].bytes = cum_size;
      }
    }
  }
  
  //Copy required data to GPU
  //TODO: assert that only last dim is tiled
  //TODO: SoA
  ops_dat_entry *item, *tmp_item;
  for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
    tmp_item = TAILQ_NEXT(item, entries);
    ops_dat dat = item->dat;

    long base_ptr = dat->base_offset;
    long end_ptr = dat->base_offset + dat->elem_size; //we calculate the last actually accessed element: -1 to dependency ranges, and +1 here
    long prod = dat->elem_size;
    for (int d = 0; d < dat->block->dims; d++) {
      //printf("%d %d-%d\n",d,dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0],dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1]);
      base_ptr += dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0] * prod * (dat->size[d]!=1);
      end_ptr  += (dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1]-1) * prod * (dat->size[d]!=1);
      prod *= dat->size[d];
    }
    if (end_ptr < base_ptr) end_ptr = base_ptr; //zero ranges
    
    //alter base_offset so that it is offset by the dependency range
    int lastdim_size = dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1]
                     - dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0];
    dat->size[dat->block->dims-1] = lastdim_size;
    //printf("Copying %s from %p+%ld to %p, size %ld. old base: %d new base %ld\n", dat->name, dat->data, base_ptr, dat->data_d, end_ptr-base_ptr, dats[dat->index].base_offset, dats[dat->index].base_offset-base_ptr);
    dat->base_offset = dats[dat->index].base_offset -  base_ptr; 
    cutilSafeCall(hipMemcpyAsync(dat->data_d, dat->data + base_ptr, end_ptr - base_ptr, hipMemcpyHostToDevice, stream_copy_up));
  }
}
void ops_finish_tile(int tile, int total_tiles, std::vector<std::vector<int> > &tiled_ranges, std::vector<std::vector<int> > &dependency_ranges) {
  //Wait for compute to finish before downloading results
  cutilSafeCall(hipStreamSynchronize(stream_compute));
  //TODO: do not copy back read-only data
  ops_dat_entry *item, *tmp_item;
  for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
    tmp_item = TAILQ_NEXT(item, entries);
    ops_dat dat = item->dat;
    //Determine data to be copied off to the CPU
    long base_ptr = dats[dat->index].base_offset;
    long end_ptr = dats[dat->index].base_offset + dat->elem_size; //we calculate the last actually accessed element: -1 to dependency ranges, and +1 here
    long prod = dat->elem_size;
    for (int d = 0; d < dat->block->dims; d++) {
      base_ptr += dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0] * prod * (dat->size[d]!=1);
      end_ptr  += (dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1]-1) * prod * (dat->size[d]!=1);
      prod *= dat->size[d];
    }
    if (end_ptr < base_ptr) end_ptr = base_ptr; //zero ranges
    cutilSafeCall(hipMemcpyAsync(dat->data + base_ptr, dat->data_d, end_ptr - base_ptr, hipMemcpyDeviceToHost, stream_copy_down));
    //printf("Copying back %s from %p+%ld to %p, size %ld. old base: %d new base %ld\n", dat->name, dat->data, base_ptr, dat->data_d, end_ptr-base_ptr, dats[dat->index].base_offset, dats[dat->index].base_offset-base_ptr);
    dat->size[dat->block->dims-1] = dats[dat->index].size[dat->block->dims-1];
    dat->base_offset = dats[dat->index].base_offset;
  }
}

