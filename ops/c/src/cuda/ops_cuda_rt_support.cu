#include "hip/hip_runtime.h"
/*
* Open source copyright declaration based on BSD open source template:
* http://www.opensource.org/licenses/bsd-license.php
*
* This file is part of the OPS distribution.
*
* Copyright (c) 2013, Mike Giles and others. Please see the AUTHORS file in
* the main source directory for a full list of copyright holders.
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
* * Redistributions of source code must retain the above copyright
* notice, this list of conditions and the following disclaimer.
* * Redistributions in binary form must reproduce the above copyright
* notice, this list of conditions and the following disclaimer in the
* documentation and/or other materials provided with the distribution.
* * The name of Mike Giles may not be used to endorse or promote products
* derived from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY Mike Giles ''AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
* WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL Mike Giles BE LIABLE FOR ANY
* DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
* (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
* LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
* ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

/** @brief ops cuda specific runtime support functions
  * @author Gihan Mudalige
  * @details Implements cuda backend runtime support functions
  */

//
// header files
//

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>

#include <ops_cuda_rt_support.h>
#include <ops_lib_core.h>

#include <vector>
#include <limits.h>

/*__global__ void copy_kernel(char *dest, char *src, int size ) {
  int tid = blockIdx.x;
  memcpy(&dest[tid],&src[tid],size);
}*/

__global__ void copy_kernel_tobuf(char *dest, char *src, int rx_s, int rx_e,
                                  int ry_s, int ry_e, int rz_s, int rz_e,
                                  int x_step, int y_step, int z_step,
                                  int size_x, int size_y, int size_z,
                                  int buf_strides_x, int buf_strides_y,
                                  int buf_strides_z, int elem_size) {

  int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
  int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
  int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

  if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
      (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
      (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

    src += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * elem_size;
    dest += ((idx_z - rz_s) * z_step * buf_strides_z +
             (idx_y - ry_s) * y_step * buf_strides_y +
             (idx_x - rx_s) * x_step * buf_strides_x) *
            elem_size;
    memcpy(dest, src, elem_size);
  }
}

__global__ void copy_kernel_frombuf(char *dest, char *src, int rx_s, int rx_e,
                                    int ry_s, int ry_e, int rz_s, int rz_e,
                                    int x_step, int y_step, int z_step,
                                    int size_x, int size_y, int size_z,
                                    int buf_strides_x, int buf_strides_y,
                                    int buf_strides_z, int elem_size) {

  int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
  int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
  int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

  if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
      (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
      (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

    dest += (idx_z * size_x * size_y + idx_y * size_x + idx_x) * elem_size;
    src += ((idx_z - rz_s) * z_step * buf_strides_z +
            (idx_y - ry_s) * y_step * buf_strides_y +
            (idx_x - rx_s) * x_step * buf_strides_x) *
           elem_size;
    memcpy(dest, src, elem_size);
  }
}

void ops_halo_copy_tobuf(char *dest, int dest_offset, ops_dat src, int rx_s,
                         int rx_e, int ry_s, int ry_e, int rz_s, int rz_e,
                         int x_step, int y_step, int z_step, int buf_strides_x,
                         int buf_strides_y, int buf_strides_z) {

  dest += dest_offset;
  int thr_x = abs(rx_s - rx_e);
  int blk_x = 1;
  if (abs(rx_s - rx_e) > 8) {
    blk_x = (thr_x - 1) / 8 + 1;
    thr_x = 8;
  }
  int thr_y = abs(ry_s - ry_e);
  int blk_y = 1;
  if (abs(ry_s - ry_e) > 8) {
    blk_y = (thr_y - 1) / 8 + 1;
    thr_y = 8;
  }
  int thr_z = abs(rz_s - rz_e);
  int blk_z = 1;
  if (abs(rz_s - rz_e) > 8) {
    blk_z = (thr_z - 1) / 8 + 1;
    thr_z = 8;
  }

  dim3 grid(blk_x, blk_y, blk_z);
  dim3 tblock(thr_x, thr_y, thr_z);
  copy_kernel_tobuf<<<grid, tblock>>>(
      dest, src->data_d, rx_s, rx_e, ry_s, ry_e, rz_s, rz_e, x_step, y_step,
      z_step, src->size[0], src->size[1], src->size[2], buf_strides_x,
      buf_strides_y, buf_strides_z, src->elem_size);

  // TODO: MPI buffers and GPUDirect
}

void ops_halo_copy_frombuf(ops_dat dest, char *src, int src_offset, int rx_s,
                           int rx_e, int ry_s, int ry_e, int rz_s, int rz_e,
                           int x_step, int y_step, int z_step,
                           int buf_strides_x, int buf_strides_y,
                           int buf_strides_z) {

  src += src_offset;
  int thr_x = abs(rx_s - rx_e);
  int blk_x = 1;
  if (abs(rx_s - rx_e) > 8) {
    blk_x = (thr_x - 1) / 8 + 1;
    thr_x = 8;
  }
  int thr_y = abs(ry_s - ry_e);
  int blk_y = 1;
  if (abs(ry_s - ry_e) > 8) {
    blk_y = (thr_y - 1) / 8 + 1;
    thr_y = 8;
  }
  int thr_z = abs(rz_s - rz_e);
  int blk_z = 1;
  if (abs(rz_s - rz_e) > 8) {
    blk_z = (thr_z - 1) / 8 + 1;
    thr_z = 8;
  }

  dim3 grid(blk_x, blk_y, blk_z);
  dim3 tblock(thr_x, thr_y, thr_z);
  copy_kernel_frombuf<<<grid, tblock>>>(
      dest->data_d, src, rx_s, rx_e, ry_s, ry_e, rz_s, rz_e, x_step, y_step,
      z_step, dest->size[0], dest->size[1], dest->size[2], buf_strides_x,
      buf_strides_y, buf_strides_z, dest->elem_size);
  dest->dirty_hd = 2;
}


__global__ void toucher(char *dat, int size, double fac) {
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  if (id < size) {
    char val = dat[id];
    if (fac == 0) val = val + 1;
    if (fac > 1) val = val * fac;
    else val = val * (char)fac;
    dat[id] = val;
  }
}
extern "C" {
void ops_touch(char *dat, int size, double fac) {
  int nthreads = 1024;
  int nblocks = (size-1)/nthreads+1;
  toucher<<<nblocks,nthreads>>>(dat,size,fac);
}
}

int mod(int a, int b)
{
    int r = a % b;
    return r < 0 ? r + b : r;
}

struct datasets {
  long bytes;
  ops_dat dat;
  int size[OPS_MAX_DIM];
  int base_offset;
  int max_width;
  int curr_slot;
  int curr_chunk[2];
  long curr_offset;
  long last_offset;
  long curr_size;
  long copy_from;
  long copy_amount;
};

std::vector<datasets> dats(0);

hipStream_t stream_copy_up = 0;
hipStream_t stream_copy_down = 0;
hipStream_t stream_compute = 0;
hipStream_t stream = 0;

//TODO: v1 seems okay 
void ops_get_offsets_deprange(long &base_ptr, long &end_ptr, ops_dat dat, std::vector<std::vector<int> > &dependency_ranges, int tile, int num_tiles, int lrf, long &delta) {

  //calculate base_offset without last used dimension
  long line_begin_offset = 0;
  long cumsize = 1;
  for (int i = 0; i < dat->block->dims-1; i++) {
    line_begin_offset +=
        dat->elem_size * cumsize * (-dat->base[i] - dat->d_m[i]); //TODO: different for MPI
    cumsize *= dats[dat->index].size[i];
  }
  line_begin_offset = dats[dat->index].base_offset - line_begin_offset;
  base_ptr = line_begin_offset; //go back to beginning of x line in 2D or last x-y plane in 3D
  end_ptr = line_begin_offset; //go back to beginning of x line in 2D or last x-y plane in 3D
  //For first n-1 dimensions, we copy all of them
  long prod = dat->elem_size;
  for (int d = 0; d < dat->block->dims-1; d++) {
    prod *= dats[dat->index].size[d];
  }
  //For the nth dimension we just copy up to where we need it
  int d = dat->block->dims -1;
  int prevrange = dependency_ranges[dat->index][mod(tile-1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 1] - dependency_ranges[dat->index][mod(tile-1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 0];
  int nextrange = dependency_ranges[dat->index][mod(tile+1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 1] - dependency_ranges[dat->index][mod(tile+1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 0];
  //Left or full - start of this tile
  if (lrf == 0 || lrf == 2 || tile == 0 || dat->size[d] == 1 || prevrange == 0)
    base_ptr += dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0] * prod;
  else //right - end of previous tile
    base_ptr += MAX(dependency_ranges[dat->index][mod(tile-1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 1],
                    dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0]) * prod;
  //Right or full - end of this tile
  if (lrf == 1 || lrf ==2 || tile == num_tiles-1 || dat->size[d] == 1 || nextrange == 0) 
    end_ptr  += dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1] * prod;
  else //left - start of next tile
    end_ptr  += MIN(dependency_ranges[dat->index][mod(tile+1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 0],
                    dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1]) * prod;

  if (dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0] 
      == dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1]) end_ptr = base_ptr; //zero dependency

  //I need to offset the beginning fo tile 0, so tile 3 (laoding to slot 0) won't bite tile 1's tail
  //extra space needed is the difference between largest tile and left range of tile 0
  if (num_tiles > 1 && tile == 0 && nextrange > 1 && dependency_ranges[dat->index][0 * 2 * OPS_MAX_DIM + 2 * d + 1] - dependency_ranges[dat->index][0 * 2 * OPS_MAX_DIM + 2 * d + 0] > 1)
    delta = (dats[dat->index].max_width - dependency_ranges[dat->index][1 * 2 * OPS_MAX_DIM + 2 * d + 0] + dependency_ranges[dat->index][0 * 2 * OPS_MAX_DIM + 2 * d + 0])*prod;
  else delta = 0;

  if (end_ptr < base_ptr) {printf("WARNING: overreaching depranges! Please check, shouldn't happen\n%s %ld-%ld, dep range: %d-%d prev %d - %d next start %d\n",dat->name, base_ptr, end_ptr, dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 0], dependency_ranges[dat->index][tile * 2 * OPS_MAX_DIM + 2 * d + 1],dependency_ranges[dat->index][mod(tile-1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 0],dependency_ranges[dat->index][mod(tile-1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 1], dependency_ranges[dat->index][mod(tile+1,num_tiles) * 2 * OPS_MAX_DIM + 2 * d + 0]); end_ptr = base_ptr;} //zero ranges
}

void ops_prepare_tile(int tile, int total_tiles, std::vector<std::vector<int> > &tiled_ranges, std::vector<std::vector<int> > &dependency_ranges) {

  if (tile == 0) {
    //TODO: v1 - this doesn't really do anything.
    cutilSafeCall(hipStreamSynchronize(stream_copy_up));
    cutilSafeCall(hipStreamSynchronize(stream_compute));

    //First time ever - initialise
    if (dats.size()==0) {
      dats.resize(dependency_ranges.size());
      for (int i = 0; i < dats.size(); i++) {
        dats[i].bytes = 0;
        dats[i].dat = NULL;
      }
      ops_dat_entry *item, *tmp_item;
      for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
        tmp_item = TAILQ_NEXT(item, entries);
        dats[item->dat->index].dat = item->dat;
        dats[item->dat->index].base_offset = item->dat->base_offset;
        memcpy(dats[item->dat->index].size, item->dat->size, sizeof(int)*OPS_MAX_DIM);
      }
      cutilSafeCall(hipStreamCreateWithFlags(&stream_copy_down,hipStreamNonBlocking));
      cutilSafeCall(hipStreamCreateWithFlags(&stream_copy_up,hipStreamNonBlocking));
      int leastPriority, greatestPriority;
      hipDeviceGetStreamPriorityRange ( &leastPriority, &greatestPriority );
      cutilSafeCall(hipStreamCreateWithPriority(&stream_compute,hipStreamNonBlocking,greatestPriority));
      stream = stream_compute;
    }

    //determine biggest dependency range for each dataset to allocate scratch memory on GPU
    ops_dat_entry *item, *tmp_item;
    for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
      tmp_item = TAILQ_NEXT(item, entries);
      int idx = item->dat->index;
      int maxsize = 0;
      for (int t = 0; t < total_tiles; t++) {
        int d = item->dat->block->dims-1; //Only for last dimension
        maxsize = MAX(maxsize,dependency_ranges[idx][t * 2 * OPS_MAX_DIM + 2 * d + 1]
                            - dependency_ranges[idx][t * 2 * OPS_MAX_DIM + 2 * d + 0]);
      }
      dats[idx].max_width = maxsize;
      //Allocate it a little larger, if not edge dat in this dim (or just unused)
      if (maxsize > 1 && dats[idx].bytes == 0) maxsize += 13;

      // total required memory
      long cum_size = item->dat->elem_size; 
      for (int d = 0; d < item->dat->block->dims-1; d++) cum_size *= item->dat->size[d];
      cum_size *= maxsize;

      //3 slots
      cum_size *= 3;
      if (cum_size > dats[idx].bytes) {
        //printf("Reallocating memory for %s: %ld->%ld\n",item->dat->name, dats[idx].bytes,cum_size);
        cutilSafeCall(hipStreamSynchronize(stream_copy_down)); //Need to make sure all previous copies finished before dealloc
        cutilSafeCall(hipFree(item->dat->data_d));
        cutilSafeCall(hipMalloc(&item->dat->data_d, cum_size));
        dats[idx].bytes = cum_size;
        dats[idx].curr_slot = 2; //last used slot, so next one is 0
        dats[idx].curr_chunk[0] = 0;
        dats[idx].curr_chunk[1] = 0;
        dats[idx].curr_offset = 0;
        dats[idx].last_offset = 0;
        dats[idx].curr_size = 0;
        dats[idx].copy_from = 0;
        dats[idx].copy_amount = 0;
      }
    }
  }
 
  //Create event that we can sync on an the end to make sure previous copies have finished 
  hipEvent_t e_copyup;
  hipEventCreate(&e_copyup);
  if (tile != 0) //TODO: v1 previous tile stack's last tile prefetching next tile stack's first tile
    hipEventRecord(e_copyup, stream_copy_up);

  //Copy required data to GPU
  //TODO: assert that only last dim is tiled
  //TODO: SoA: set dat->size[dat->block->dims-1]
  ops_dat_entry *item, *tmp_item;
  if (tile == 0) { //TODO: v1, will need to check if it was uploaded predictively okay
    hipStreamSynchronize(stream_copy_down);
    for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
      tmp_item = TAILQ_NEXT(item, entries);
      ops_dat dat = item->dat;
      int idx = dat->index;

      //Determine data to be copied up to the GPU
      long base_ptr, end_ptr, delta;
      ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, tile, total_tiles, 2, delta); //Full
      printf("Tile 0 Copying %s from %p+%ld to %p+%ld (%p-%p), size %ld\n", dat->name, dat->data, base_ptr, dat->data_d,delta, dat->data_d, dat->data_d + dats[idx].bytes, end_ptr-base_ptr);
      cutilSafeCall(hipMemcpyAsync(dat->data_d+delta, dat->data + base_ptr, end_ptr - base_ptr, hipMemcpyHostToDevice, stream_copy_up));
      dats[idx].curr_slot = 0;
      dats[idx].curr_offset = delta;
      dats[idx].last_offset = 0;
      dats[idx].curr_size = end_ptr-base_ptr;
      dats[idx].curr_chunk[0] = dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0];
      dats[idx].curr_chunk[1] = dependency_ranges[idx][tile * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1];
      dats[idx].copy_from = 0;
      dats[idx].copy_amount = 0;
    }
    hipEventRecord(e_copyup, stream_copy_up);
  }
  if (tile != total_tiles - 1) { //TODO: v1 upload next tile - leave a few rows spare if next tile's dependency range is bigger
    for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
      tmp_item = TAILQ_NEXT(item, entries);
      ops_dat dat = item->dat;
      int idx = dat->index;

      //For edge thats I do not need to upload again
      //TODO: v1
      if (dat->size[dat->block->dims-1] == 1) continue;

      //Upload next tile
      long base_ptr, end_ptr, delta;
      ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, mod(tile+1,total_tiles), total_tiles, 1, delta); //Right
      if (dats[idx].curr_slot < 2) {
        printf("Prefetching tile %d to slot %d Copying %s from %p+%ld to %p+%ld (%p-%p), size %ld\n", tile+1, dats[idx].curr_slot+1, dat->name, dat->data, base_ptr, dat->data_d, dats[idx].curr_offset + dats[idx].curr_size, dat->data_d, dat->data_d + dats[idx].bytes, end_ptr-base_ptr);
        cutilSafeCall(hipMemcpyAsync(dat->data_d + dats[idx].curr_offset + dats[idx].curr_size,
                dat->data + base_ptr, end_ptr - base_ptr, hipMemcpyHostToDevice, stream_copy_up));
        dats[idx].curr_slot++; //Smaller than two, so just increment
        dats[idx].last_offset = dats[idx].curr_offset;
        dats[idx].curr_offset += dats[idx].curr_size; //end of previous
        dats[idx].curr_size = end_ptr-base_ptr;
        dats[idx].copy_from = 0;
        dats[idx].copy_amount = 0;
      } else { //Going to first slot, need extra offset, and copy of previous tile's overlapping dependency range
        //Compute Full range, then right begin - full begin is the extra offset
        long base_ptr2, end_ptr2;
        ops_get_offsets_deprange(base_ptr2, end_ptr2, dat, dependency_ranges, mod(tile+1,total_tiles), total_tiles, 2, delta); //Full
        long extra_offset = base_ptr - base_ptr2;
        printf("Prefetching tile %d to slot %d Copying %s from %p+%ld to %p+%ld (%p-%p), size %ld\n", tile+1, 0, dat->name, dat->data, base_ptr, dat->data_d, extra_offset, dat->data_d, dat->data_d + dats[idx].bytes, end_ptr-base_ptr);
        cutilSafeCall(hipMemcpyAsync(dat->data_d + extra_offset,
                dat->data + base_ptr, end_ptr - base_ptr, hipMemcpyHostToDevice, stream_copy_up));
        dats[idx].copy_from = dats[idx].curr_offset + dats[idx].curr_size - extra_offset;
        dats[idx].copy_amount = extra_offset;
        dats[idx].curr_slot = 0;
        dats[idx].last_offset = dats[idx].curr_offset;
        dats[idx].curr_offset = 0;
        dats[idx].curr_size = end_ptr - base_ptr2; //specify full size with extra on left

        //Do an extra check for potential overlap
        ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, 0, total_tiles, 2, delta);
        if (dats[idx].curr_size + extra_offset > delta + end_ptr-base_ptr) printf("Warning - potential race condition %s delta %ld (%d-%d)\n",dat->name, delta,dependency_ranges[dat->index][0 * 2 * OPS_MAX_DIM + 2 * (dat->block->dims-1) + 1], dependency_ranges[dat->index][1 * 2 * OPS_MAX_DIM + 2 * (dat->block->dims-1) + 0]);
      }
      dats[idx].curr_chunk[0] = dependency_ranges[dat->index][mod(tile+1,total_tiles) * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 0];
      dats[idx].curr_chunk[1] = dependency_ranges[dat->index][mod(tile+1,total_tiles) * 2 * OPS_MAX_DIM + 2 * (dat->block->dims - 1) + 1];
    }
  }

  //alter base_offset so that it is offset by the dependency range for the current tile
  for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
    tmp_item = TAILQ_NEXT(item, entries);
    ops_dat dat = item->dat;
    if ((tile < total_tiles-1 && dats[dat->index].curr_slot == 1) || (tile == total_tiles-1 && dats[dat->index].curr_slot == 0) || dat->size[dat->block->dims-1] == 1) { //TODO: v1 upload next handling
      long base_ptr, end_ptr, delta;
      ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, tile, total_tiles, 2, delta); //Full
      //printf("New base offset for %s: %ld->%ld\n",dat->name, dat->base_offset, dats[dat->index].base_offset -  base_ptr);
      dat->base_offset = dats[dat->index].base_offset -  base_ptr + delta; //TODO: v1 little extra for first tile for safety
    }
  }

  //Before actually starting the computations, make sure previous copies up finished
  hipEventSynchronize(e_copyup);
  hipEventDestroy(e_copyup);
}
void ops_finish_tile(int tile, int total_tiles, std::vector<std::vector<int> > &tiled_ranges, std::vector<std::vector<int> > &dependency_ranges) {
  hipEvent_t e_copydown;
  hipEventCreate(&e_copydown);
  hipEventRecord(e_copydown, stream_copy_down);

  //Wait for compute to finish before downloading results
  cutilSafeCall(hipStreamSynchronize(stream_compute));

  //TODO: do not copy back read-only data
  ops_dat_entry *item, *tmp_item;
  for (item = TAILQ_FIRST(&OPS_dat_list); item != NULL; item = tmp_item) {
    tmp_item = TAILQ_NEXT(item, entries);
    ops_dat dat = item->dat;
    int idx = dat->index;
    //Restore properties
    if (tile == total_tiles-1)
      dat->base_offset = dats[dat->index].base_offset;

    //Skip edge dats, those are managed by the first tile
    if (tile > 0 && dat->size[dat->block->dims-1] == 1) continue;

    //Copy over the right edge of this tile in the last slot, to the left of the first slot
    if (dats[idx].curr_slot == 0 && dats[idx].copy_amount > 0) {
      cutilSafeCall(hipMemcpyAsync(dat->data_d, dat->data_d+dats[idx].copy_from, dats[idx].copy_amount, hipMemcpyDeviceToDevice, stream_compute));
      dats[idx].copy_from = 0;
      dats[idx].copy_amount = 0;
    }
    //Determine data to be copied off to the CPU
    long base_ptr, end_ptr, delta;
    ops_get_offsets_deprange(base_ptr, end_ptr, dat, dependency_ranges, tile, total_tiles, (tile == total_tiles-1 ? 2 : 0),delta); //Left or full if last tile
    //Where the data being processed starts
    long base_ptr_gpu = tile == total_tiles-1 ? dats[idx].curr_offset : dats[idx].last_offset;
    //if we are not downloading from slot 0, then last_offset does not contain the left part of the tile
    if (!(tile < total_tiles-1 && dats[idx].curr_slot == 1) && !(tile == total_tiles-1 && dats[idx].curr_slot==0)) {
      long base_ptr2, end_ptr2;
      ops_get_offsets_deprange(base_ptr2, end_ptr2, dat, dependency_ranges, tile, total_tiles, 1, delta); //Right
      base_ptr_gpu -= (base_ptr2-base_ptr);
    }
    printf("Tile %d copying back %s to %p+%ld from %p+%ld, size %ld\n", tile, dat->name, dat->data, base_ptr, dat->data_d, base_ptr_gpu,end_ptr-base_ptr);
    cutilSafeCall(hipMemcpyAsync(dat->data + base_ptr, dat->data_d+base_ptr_gpu, end_ptr - base_ptr, hipMemcpyDeviceToHost, stream_copy_down));
  }

  //Wait for previous round of copies to finish
  hipEventSynchronize(e_copydown);
  hipEventDestroy(e_copydown);
  if (((double*)dats[27].dat->data)[202] == 0.0 && tile > 1) {printf("Zero detected for tile %d\n",tile-1);}
//Problem if there is no sync on stream_copy_up here - i.e. ths unload overlaps with next load
}

